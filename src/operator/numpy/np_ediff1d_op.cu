/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_ediff1d_op.cu
 * \brief GPU implementation of numpy-compatible ediff1d operator
 */

#include "./np_ediff1d_op-inl.h"

namespace mxnet {
namespace op {

void EDiff1DForwardGPU(const nnvm::NodeAttrs& attrs,
                       const OpContext& ctx,
                       const std::vector<TBlob>& inputs,
                       const std::vector<OpReqType>& req,
                       const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mxnet_op;
  CHECK_GE(inputs.size(), 1U);
  CHECK_LE(inputs.size(), 3U);
  CHECK_EQ(req.size(), 1U);
  CHECK_EQ(outputs.size(), 1U);
  const TBlob& in_data = inputs[0];
  const TBlob& out_data = outputs[0];
  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  MSHADOW_TYPE_SWITCH(out_data.type_flag_, DType, {
    const EDiff1DParam& param = nnvm::get<EDiff1DParam>(attrs.parsed);
    size_t padding = 0;
    size_t in_size = (in_data.Size() > 0)? in_data.Size() - 1: 0;
    index_t idx = 1;  // used to index the rest of input arrays

    if (param.to_begin_arr_given) {
      // if the `to_begin` parameter is an array, copy its values to the beginning of the out array
      CUDA_CALL(hipMemcpyAsync(out_data.dptr<DType>(), inputs[idx].dptr<DType>(),
                                inputs[idx].Size() * sizeof(DType), hipMemcpyDeviceToHost,
                                mshadow::Stream<gpu>::GetStream(s)));
      padding += inputs[idx].Size();
      idx += 1;
    } else if (param.to_begin_scalar.has_value()) {
      // if the `to_begin` parameter is a scalar, directly assign its value
      out_data.dptr<DType>()[0] = param.to_begin_scalar.value();
      padding += 1;
    }

    if (param.to_end_arr_given) {
      // if the `to_end` parameter is an array, copy its values to the end of the out array
      CUDA_CALL(hipMemcpyAsync(out_data.dptr<DType>() + padding + in_size,
                                inputs[idx].dptr<DType>(),
                                inputs[idx].Size() * sizeof(DType),
                                hipMemcpyDeviceToHost,
                                mshadow::Stream<gpu>::GetStream(s)));
    } else if (param.to_end_scalar.has_value()) {
      // if the `to_end` parameter is a scalar, directly assign its value
      out_data.dptr<DType>()[padding + in_size] = param.to_end_scalar.value();
    }

    MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
      Kernel<ediff1d_forward<req_type>, gpu>::Launch(
        s, in_size, out_data.dptr<DType>(), in_data.dptr<DType>(), padding);
    });
  });
}

void EDiff1DBackwardGPU(const nnvm::NodeAttrs& attrs, const OpContext& ctx,
                        const std::vector<TBlob>& inputs,
                        const std::vector<OpReqType>& req,
                        const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  using namespace mxnet_op;
  CHECK_GE(inputs.size(), 2U);
  CHECK_LE(inputs.size(), 4U);
  CHECK_GE(outputs.size(), 1U);
  CHECK_LE(outputs.size(), 3U);
  CHECK_EQ(req.size(), outputs.size());

  mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
  const EDiff1DParam& param = nnvm::get<EDiff1DParam>(attrs.parsed);

  const TBlob& ograd = inputs[0];
  const TBlob& input = inputs[1];
  const TBlob& igrad = outputs[0];
  size_t in_size = (input.Size() > 0)? input.Size() - 1: 0;

  MSHADOW_REAL_TYPE_SWITCH(ograd.type_flag_, DType, {
    MXNET_ASSIGN_REQ_SWITCH(req[0], req_type, {
      size_t padding = 0;
      index_t idx = 1;  // start from the second argument of `outputs`
      if (param.to_begin_arr_given) {
        CUDA_CALL(hipMemcpyAsync(outputs[idx].dptr<DType>(),
                                  ograd.dptr<DType>(),
                                  outputs[idx].Size() * sizeof(DType),
                                  hipMemcpyDeviceToHost,
                                  mshadow::Stream<gpu>::GetStream(s)));
        padding += outputs[idx].Size();
        idx += 1;
      } else if (param.to_begin_scalar.has_value()) {
        padding += 1;
      }

      if (param.to_end_arr_given) {
        CUDA_CALL(hipMemcpyAsync(outputs[idx].dptr<DType>(),
                                  ograd.dptr<DType>()+ in_size + padding,
                                  outputs[idx].Size() * sizeof(DType),
                                  hipMemcpyDeviceToHost,
                                  mshadow::Stream<gpu>::GetStream(s)));
      }

      if (input.Size() == 0) return;
      if (input.Size() == 1) {
        Kernel<set_to_val<DType>, gpu>::Launch(s, 1, igrad.dptr<DType>(), 0);
      } else {
        Kernel<ediff1d_backward_arr<req_type>, gpu>::Launch(
          s, igrad.Size(), igrad.dptr<DType>(),
          input.dptr<DType>(), ograd.dptr<DType>(),
          padding, igrad.Size());
      }
    });
  });
}

NNVM_REGISTER_OP(_npi_ediff1d)
.set_attr<FCompute>("FCompute<gpu>", EDiff1DForwardGPU);

NNVM_REGISTER_OP(_npi_backward_ediff1d)
.set_attr<FCompute>("FCompute<gpu>", EDiff1DBackwardGPU);

}  // namespace op
}  // namespace mxnet
